/*
   STREAM benchmark implementation in CUDA.

COPY:       a(i) = b(i)
SCALE:      a(i) = q*b(i)
SUM:        a(i) = b(i) + c(i)
TRIAD:      a(i) = b(i) + q*c(i)

It measures the memory system on the device.
The implementation is in double precision.

Code based on the code developed by John D. McCalpin
http://www.cs.virginia.edu/stream/FTP/Code/stream.c

Written by: Massimiliano Fatica, NVIDIA Corporation

Further modifications by: Ben Cumming, CSCS; Andreas Herten (JSC/FZJ); Sebastian Achilles (JSC/FZJ)
 */

#ifdef NTIMES
#if NTIMES <= 1
#   define NTIMES  20
#endif
#endif
#ifndef NTIMES
#   define NTIMES  20
#endif


#include <hip/hip_runtime.h>
#include <string>
#include <vector>

#include <stdio.h>
#include <float.h>
#include <limits.h>
// #include <unistd.h>
#include <getopt.h>

#include <chrono>

# ifndef MIN
# define MIN(x,y) ((x)<(y)?(x):(y))
# endif
# ifndef MAX
# define MAX(x,y) ((x)>(y)?(x):(y))
# endif

typedef double real;

static double   avgtime[4] = {0}, maxtime[4] = {0},
                mintime[4] = {FLT_MAX,FLT_MAX,FLT_MAX,FLT_MAX};


void print_help()
{
  printf(
      "Usage: stream [-s] [-c [-f]] [-n <elements>] [-b <blocksize>]\n\n"
      "  -s, --si\n"
      "        Print results in SI units (by default IEC units are used)\n\n"
      "  -c, --csv\n"
      "        Print results CSV formatted\n\n"
      "  -f, --full\n"
      "        Print all results in CSV\n\n"
      "  -t, --title\n"
      "        Print CSV header\n\n"
      "  -n <elements>, --nelements <element>\n"
      "        Put <elements> values in the arrays\n"
      "        (default: 1<<26)\n\n"
      "  -b <blocksize>, --blocksize <blocksize>\n"
      "        Use <blocksize> as the number of threads in each block\n"
      "        (default: 192)\n"
      );
}

void parse_options(int argc, char** argv, bool& SI, bool& CSV, bool& CSV_full, bool& CSV_header, int& N, int& blockSize)
{
  // Default values
  SI = false;
  CSV = false;
  CSV_full = false;
  CSV_header = false;
  N = 1<<26;
  blockSize = 192;

  static struct option long_options[] = {
    {"si",        no_argument,       0,  's' },
    {"csv",       no_argument,       0,  'c' },
    {"full",      no_argument,       0,  'f' },
    {"title",     no_argument,       0,  't' },
    {"nelements", required_argument, 0,  'n' },
    {"blocksize", required_argument, 0,  'b' },
    {"help",      no_argument,       0,  'h' },
    {0,           0,                 0,  0   }
  };
  int c;
  int option_index = 0;
  while ((c = getopt_long(argc, argv, "scftn:b:h", long_options, &option_index)) != -1)
    switch (c)
    {
      case 's':
        SI = true;
        break;
      case 'c':
        CSV = true;
        break;
      case 'f':
        CSV_full = true;
        break;
      case 't':
        CSV_header = true;
        break;
      case 'n':
        N = std::atoi(optarg);
        break;
      case 'b':
        blockSize = std::atoi(optarg);
        break;
      case 'h':
        print_help();
        std::exit(0);
        break;
      default:
        print_help();
        std::exit(1);
    }
}

  template <typename T>
__global__ void set_array(T * __restrict__ const a, T value, int len)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len)
    a[idx] = value;
}

  template <typename T>
__global__ void STREAM_Copy(T const * __restrict__ const a, T * __restrict__ const b, int len)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len)
    b[idx] = a[idx];
}

  template <typename T>
__global__ void STREAM_Scale(T const * __restrict__ const a, T * __restrict__ const b, T scale,  int len)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len)
    b[idx] = scale * a[idx];
}

  template <typename T>
__global__ void STREAM_Add(T const * __restrict__ const a, T const * __restrict__ const b, T * __restrict__ const c, int len)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len)
    c[idx] = a[idx] + b[idx];
}

  template <typename T>
__global__ void STREAM_Triad(T const * __restrict__ a, T const * __restrict__ b, T * __restrict__ const c, T scalar, int len)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len)
    c[idx] = a[idx] + scalar * b[idx];
}

int main(int argc, char** argv)
{
  real *d_a, *d_b, *d_c;
  int j,k;
  double times[4][NTIMES];
  real scalar;
  std::chrono::steady_clock::time_point start_time, end_time;
  std::vector<std::string> label{"Copy:      ", "Scale:     ", "Add:       ", "Triad:     "};

  // Parse arguments
  bool SI, CSV, CSV_full, CSV_header;
  int N, blockSize;
  parse_options(argc, argv, SI, CSV, CSV_full, CSV_header, N, blockSize);

  if (!CSV) {
    printf("STREAM Benchmark implementation in CUDA\n");
    printf("Array size (%s precision) = %7.2f MB\n", sizeof(double)==sizeof(real)?"double":"single", double(N)*double(sizeof(real))/1.e6);
  }

  /* Allocate memory on device */
#ifdef USE_HOST
  d_a=(real*)malloc(sizeof(real)*N);
  d_b=(real*)malloc(sizeof(real)*N);
  d_c=(real*)malloc(sizeof(real)*N);
#elif defined(ZERO_COPY)
  real *h_a, *h_b, *h_c;
  cudaHostAlloc((void **) &h_a, sizeof(real)*N, cudaHostAllocMapped);
  cudaHostAlloc((void **) &h_b, sizeof(real)*N, cudaHostAllocMapped);
  cudaHostAlloc((void **) &h_c, sizeof(real)*N, cudaHostAllocMapped);

  // these compiles fine but don't run correctly.
  //h_a=(real*)malloc(sizeof(real)*N);
  //h_b=(real*)malloc(sizeof(real)*N);
  //h_c=(real*)malloc(sizeof(real)*N);

  cudaHostGetDevicePointer((void **) &d_a, (void *) h_a, 0);
  cudaHostGetDevicePointer((void **) &d_b, (void *) h_b, 0);
  cudaHostGetDevicePointer((void **) &d_c, (void *) h_c, 0);
#else 
  hipMalloc((void**)&d_a, sizeof(real)*N);
  hipMalloc((void**)&d_b, sizeof(real)*N);
  hipMalloc((void**)&d_c, sizeof(real)*N);
#endif

  /* Compute execution configuration */
  dim3 dimBlock(blockSize);
  dim3 dimGrid(N/dimBlock.x );
  if( N % dimBlock.x != 0 ) dimGrid.x+=1;

  if (!CSV) {
    printf("Using %d threads per block, %d blocks\n",dimBlock.x,dimGrid.x);

    if (SI)
      printf("Output in SI units (KB = 1000 B)\n");
    else
      printf("Output in IEC units (KiB = 1024 B)\n");
  }

  /* Initialize memory on the device */
  set_array<real><<<dimGrid,dimBlock>>>(d_a, 2.f, N);
  set_array<real><<<dimGrid,dimBlock>>>(d_b, .5f, N);
  set_array<real><<<dimGrid,dimBlock>>>(d_c, .5f, N);

  /*  --- MAIN LOOP --- repeat test cases NTIMES times --- */

  scalar=3.0f;
  for (k=0; k<NTIMES; k++)
  {
    start_time = std::chrono::steady_clock::now();
    STREAM_Copy<real><<<dimGrid,dimBlock>>>(d_a, d_c, N);
    hipDeviceSynchronize();
    end_time = std::chrono::steady_clock::now();
    times[0][k] = std::chrono::duration_cast<std::chrono::duration<double>>(end_time - start_time).count();

    start_time = std::chrono::steady_clock::now();
    STREAM_Scale<real><<<dimGrid,dimBlock>>>(d_b, d_c, scalar,  N);
    hipDeviceSynchronize();
    end_time = std::chrono::steady_clock::now();
    times[1][k] = std::chrono::duration_cast<std::chrono::duration<double>>(end_time - start_time).count();

    start_time = std::chrono::steady_clock::now();
    STREAM_Add<real><<<dimGrid,dimBlock>>>(d_a, d_b, d_c,  N);
    hipDeviceSynchronize();
    end_time = std::chrono::steady_clock::now();
    times[2][k] = std::chrono::duration_cast<std::chrono::duration<double>>(end_time - start_time).count();

    start_time = std::chrono::steady_clock::now();
    STREAM_Triad<real><<<dimGrid,dimBlock>>>(d_b, d_c, d_a, scalar,  N);
    hipDeviceSynchronize();
    end_time = std::chrono::steady_clock::now();
    times[3][k] = std::chrono::duration_cast<std::chrono::duration<double>>(end_time - start_time).count();
  }

  /*  --- SUMMARY --- */

  for (k=1; k<NTIMES; k++) /* note -- skip first iteration */
  {
    for (j=0; j<4; j++)
    {
      avgtime[j] = avgtime[j] + times[j][k];
      mintime[j] = MIN(mintime[j], times[j][k]);
      maxtime[j] = MAX(maxtime[j], times[j][k]);
    }
  }
  for (j=0; j<4; j++)
    avgtime[j] = avgtime[j]/(double)(NTIMES-1);

  double bytes[4] = {
    2 * sizeof(real) * (double)N,
    2 * sizeof(real) * (double)N,
    3 * sizeof(real) * (double)N,
    3 * sizeof(real) * (double)N
  };

  // Use right units
  const double G = SI ? 1.e9 : static_cast<double>(1<<30);
  std::string gbpersec = SI ? "GB/s" : "GiB/s";

  if (!CSV) {
    printf("\nFunction      Rate %s  Avg time(s)  Min time(s)  Max time(s)\n", gbpersec.c_str() );
    printf("-----------------------------------------------------------------\n");
    for (j=0; j<4; j++) {
      printf("%s%11.2f     %11.8f  %11.8f  %11.8f\n", label[j].c_str(),
          bytes[j]/mintime[j] / G,
          avgtime[j],
          mintime[j],
          maxtime[j]);
    }
  } else {
    if (CSV_full) {
      if (CSV_header)
        printf("Copy (Max) / %s, Copy (Min) / %s, Copy (Avg) / %s, Scale (Max) / %s, Scale (Min) / %s, Scale (Avg) / %s, Add (Max) / %s, Add (Min) / %s, Add (Avg) / %s, Triad (Max) / %s, Triad (Min) / %s, Triad (Avg) / %s\n",
            gbpersec.c_str(), gbpersec.c_str(), gbpersec.c_str(),
            gbpersec.c_str(), gbpersec.c_str(), gbpersec.c_str(),
            gbpersec.c_str(), gbpersec.c_str(), gbpersec.c_str(),
            gbpersec.c_str(), gbpersec.c_str(), gbpersec.c_str()
            );
      printf(
          "%0.4f,%0.4f,%0.4f,%0.4f,%0.4f,%0.4f,%0.4f,%0.4f,%0.4f,%0.4f,%0.4f,%0.4f\n",
          bytes[0]/mintime[0] / G, bytes[0]/maxtime[0] / G, bytes[0]/(avgtime[0])/ G,
          bytes[1]/mintime[1] / G, bytes[1]/maxtime[1] / G, bytes[1]/(avgtime[1]) / G,
          bytes[2]/mintime[2] / G, bytes[2]/maxtime[2] / G, bytes[2]/(avgtime[2]) / G,
          bytes[3]/mintime[3] / G, bytes[3]/maxtime[3] / G, bytes[3]/(avgtime[3]) / G
          );
    }
    else {
      if (CSV_header)
        printf("Copy (Max) / %s, Scale (Max) / %s, Add (Max) / %s, Triad (Max) / %s\n", gbpersec.c_str(), gbpersec.c_str(), gbpersec.c_str(), gbpersec.c_str());
      printf(
          "%0.4f,%0.4f,%0.4f,%0.4f\n",
          bytes[0]/mintime[0] / G,
          bytes[1]/mintime[1] / G,
          bytes[2]/mintime[2] / G,
          bytes[3]/mintime[3] / G
          );
    }
  }


  /* Free memory on device */
#ifdef USE_HOST
  free(d_a);
  free(d_b);
  free(d_c);
#elif defined(ZERO_COPY)
  cudaFree(d_a);
  cudaFree(d_b);
  cudaFree(d_c);
  cudaFreeHost(h_a);
  cudaFreeHost(h_b);
  cudaFreeHost(h_c);
//  free(h_a);
//  free(h_b);
//  free(h_c);
#else
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
#endif
}

